#include <stdio.h>
#include "hip/hip_runtime.h"


__global__ void matrixMulRow(int *d_mat1, int *d_mat2, int *d_ans, int m, int n) {
	int tid=blockIdx.x;
    int sum = 0;
    for(int j=0; j<m; j++) {
        for(int i=0; i<n; i++)
            sum += d_mat1[tid*m+i] * d_mat2[j+n*i];
        d_ans[tid*m+j] = sum;
        sum = 0;
    }
}

__global__ void matrixMulCol(int *d_mat1, int *d_mat2, int *d_ans, int m, int n) {
	int tid=blockIdx.x;
    int sum = 0;
    for(int j=0; j<m; j++) {
        for(int i=0; i<n; i++)
            sum += d_mat1[j*m+i] * d_mat2[tid+n*i];
        d_ans[tid+n*j] = sum;
        sum = 0;
    }
}

__global__ void matrixMulEle(int *d_mat1, int *d_mat2, int *d_ans, int m, int n) {
	int tid=blockIdx.x;
    int row = tid/n, col = tid%n;
    int sum = 0;
	for(int i=0; i<n; i++)
        sum += d_mat1[n*row+i] * d_mat2[n*i+col];
    d_ans[tid] = sum;
}

int main(void) {
	int m, n;

	printf("Enter the row and col size of the matrix: ");
	scanf("%d %d", &m, &n);

	int *mat1, *mat2, *ans;
	int *d_mat1, *d_mat2, *d_ans;

	mat1 = (int*)malloc(m*n*sizeof(int));
	mat2 = (int*)malloc(m*n*sizeof(int));
	ans = (int*)malloc(m*n*sizeof(int));

	hipMalloc((void **)&d_mat1, m*n*sizeof(int));
	hipMalloc((void **)&d_mat2, m*n*sizeof(int));
	hipMalloc((void **)&d_ans, m*n*sizeof(int));

    printf("Enter the elements of the matrix1: ");
		for(int i=0; i<m*n; i++)
			scanf("%d", &mat1[i]);
	printf("Enter the elements of the matrix2: ");
		for(int i=0; i<m*n; i++)
			scanf("%d", &mat2[i]);

	hipMemcpy(d_mat1, mat1, m*n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_mat2, mat2, m*n*sizeof(int), hipMemcpyHostToDevice);

	matrixMulRow<<<m, 1>>>(d_mat1, d_mat2, d_ans, m, n);

	hipMemcpy(ans, d_ans, m*n*sizeof(int), hipMemcpyDeviceToHost);
	
	printf("Final matrix is:\n");
	for(int i=0; i<m*n; i++) {
		if(i%n == 0)
			printf("\n");
		printf("%d ", ans[i]);
	}

	matrixMulCol<<<m, 1>>>(d_mat1, d_mat2, d_ans, m, n);

	hipMemcpy(ans, d_ans, m*n*sizeof(int), hipMemcpyDeviceToHost);
	
	printf("Final matrix is:\n");
	for(int i=0; i<m*n; i++) {
		if(i%n == 0)
			printf("\n");
		printf("%d ", ans[i]);
	}

	matrixMulEle<<<m*n, 1>>>(d_mat1, d_mat2, d_ans, m, n);

	hipMemcpy(ans, d_ans, m*n*sizeof(int), hipMemcpyDeviceToHost);
	
	printf("Final matrix is:\n");
	for(int i=0; i<m*n; i++) {
		if(i%n == 0)
			printf("\n");
		printf("%d ", ans[i]);
	}
	
	hipFree(d_mat1);
	hipFree(d_mat2);
	hipFree(d_ans);

	return 0;
}
